#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dotProd_kernel_i32(int *a, int *b, long long *c, unsigned long long size) {
    __shared__ int cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    int tmp = 0;
    while (tid < n) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = tmp;

    __syncthreads();

    // do reduction in shared mem
    int i = blockDim.x >> 1;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i >>= 1;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

__global__ void dotProd_kernel_f64(double *a, double *b, long double *c, unsigned long long n) {
    __shared__ double cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    double tmp = 0;
    while (tid < n) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = tmp;

    __syncthreads();

    // do reduction in shared mem
    int i = blockDim.x >> 1;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i >>= 1;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int dotproduct_cuda_i32(int *a, int *b, long long *result, unsigned long long size) {
    int *d_a, *d_b;
    long long *d_c, *partial_results;
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    if (blocksPerGrid > 65535) blocksPerGrid = 65535;
    
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, blocksPerGrid * sizeof(long long));
    
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    
    partial_results = (long long*)malloc(blocksPerGrid * sizeof(long long));
    
    dotProd_kernel_i32<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipDeviceSynchronize();
    hipMemcpy(partial_results, d_c, blocksPerGrid * sizeof(long long), hipMemcpyDeviceToHost);
    
    // reduce the partial results on the host
    *result = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        *result += partial_results[i];
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(partial_results);
    
    return 0;
}

int dotproduct_cuda_f64(double *a, double *b, long double *result, unsigned long long size) {
    double *d_a, *d_b;
    long double *d_c, *partial_results;
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    if (blocksPerGrid > 65535) blocksPerGrid = 65535;
    
    hipMalloc((void**)&d_a, size * sizeof(double));
    hipMalloc((void**)&d_b, size * sizeof(double));
    hipMalloc((void**)&d_c, blocksPerGrid * sizeof(long double));
    
    hipMemcpy(d_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    
    partial_results = (long double*)malloc(blocksPerGrid * sizeof(long double));
    
    dotProd_kernel_f64<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipDeviceSynchronize();
    hipMemcpy(partial_results, d_c, blocksPerGrid * sizeof(long double), hipMemcpyDeviceToHost);
    
    // reduce the partial results on the host
    *result = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        *result += partial_results[i];
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(partial_results);
    
    return 0;
}

int main(int argc, char *argv[]) {
    // Example usage
    int a[256], b[256];
    long long result_i32;
    long double result_f64;

    for (int i = 0; i < 256; i++) {
        a[i] = i;
        b[i] = i;
    }

    dotproduct_cuda_i32(a, b, &result_i32, 256);
    printf("Dot product (int): %lld\n", result_i32);

    double a_f64[256], b_f64[256];
    for (int i = 0; i < 256; i++) {
        a_f64[i] = (double)i;
        b_f64[i] = (double)i;
    }

    dotproduct_cuda_f64(a_f64, b_f64, &result_f64, 256);
    printf("Dot product (double): %Lf\n", result_f64);

    return 0;
}