
#include <hip/hip_runtime.h>
__global__ void dotProd_kernel_i32(int *a, int *b, int *c, int n) {
    __shared__ int cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    int tmp = 0;
    while (tid < n) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = tmp;

    __syncthreads();

    // do reduction in shared mem
    int i = blockDim.x >> 1;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i >>= 1;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

__global__ void dotProd_kernel_f32(float *a, float *b, float *c, int n) {
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float tmp = 0;
    while (tid < n) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = tmp;

    __syncthreads();

    // do reduction in shared mem
    int i = blockDim.x >> 1;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i >>= 1;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}